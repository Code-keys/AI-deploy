#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include "mish.h"

namespace nvinfer1
{
    MishPlugin::MishPlugin(const int cudaThread) : thread_count_(cudaThread)
    {
    }
    
    MishPlugin::~MishPlugin()
    {
    }
    
    // create the plugin at runtime from a byte stream
    MishPlugin::MishPlugin(const void* data, size_t length)
    {
        assert(length == sizeof(input_size_));
        input_size_ = *reinterpret_cast<const int*>(data);
    }

    void MishPlugin::serialize(void* buffer)
    {
        *reinterpret_cast<int*>(buffer) = input_size_;
    }
    
    size_t MishPlugin::getSerializationSize()
    {  
        return sizeof(input_size_);
    }

    int MishPlugin::initialize()
    { 
        return 0;
    }
    
    Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        assert(nbInputDims == 1);
        assert(index == 0);
        input_size_ = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2];
        // Output dimensions
        return DimsCHW(inputs[0].d[0], inputs[0].d[1], inputs[0].d[2]);
    }

    __device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}

    __device__ float softplus_kernel(float x, float threshold = 20) {
        if (x > threshold) return x;                // too large
        else if (x < -threshold) return expf(x);    // too small
        return logf(expf(x) + 1);
    }

    __global__ void mish_kernel(const float *input, float *output, int num_elem) {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        //float t = exp(input[idx]);
        //if (input[idx] > 20.0) {
        //    t *= t;
        //    output[idx] = (t - 1.0) / (t + 1.0);
        //} else {
        //    float tt = t * t;
        //    output[idx] = (tt + 2.0 * t) / (tt + 2.0 * t + 2.0);
        //}
        //output[idx] *= input[idx];
        output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
    }

    void MishPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
        int block_size = thread_count_;
        int grid_size = (input_size_ * batchSize + block_size - 1) / block_size;
        mish_kernel<<<grid_size, block_size>>>(inputs[0], output, input_size_ * batchSize);
    }


    int MishPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }
}

